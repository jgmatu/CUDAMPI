#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// int pos = threadIdx.x + blockIdx.x * blockDim.x;

#define BLOCKS 8
#define THREADS 32

__global__ void kernelVacio( void ) {
      if (threadIdx.x < 10) {
            printf("Data: %s Id Thread: %d Id block : %d Num threads block : %d\n", "helloWorld!", threadIdx.x, blockIdx.x, blockDim.x);
      }
}

int main( void ) {
      kernelVacio<<<BLOCKS, THREADS>>>(); //kernel launch con grid mínimo
      hipDeviceSynchronize();
      return 0;
}
