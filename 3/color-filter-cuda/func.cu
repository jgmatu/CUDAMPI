//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <algorithm>    // std::max
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

typedef unsigned char uchar;
#define FILTER_WIDTH 3
#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__constant__ float filtro[FILTER_WIDTH*FILTER_WIDTH];

__global__
void box_filter(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* __restrict__ filter, const int filterWidth)
{
	//Aplicar el filtro a cada pixel de la imagen

  // NOTA: Que un thread tenga una posici�n correcta en 2D no quiere decir que al aplicar el filtro
  // los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.
}



//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{

	//Separar la imagen en sus canales RGB
}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
	//Recombinar los canales RGB en una sola imagen
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
  //Reservar memoria para los 3 canales y el filtro de color
}


void create_filter(float **h_filter, int *filterWidth){

	//crear el filtro se que va a aplicar (en CPU) y almacenar su tamaño
}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redFiltered, 
                        unsigned char *d_greenFiltered, 
                        unsigned char *d_blueFiltered,
                        const int filterWidth)
{
	//Separar la imagen en sus canales rgb
	//aplicar el filtro a cada canal
	//volver a juntar la imagen

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
