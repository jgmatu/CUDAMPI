#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 16
#define BLOCKS 8


__global__ void __add__(int *array, int *size) {
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      if (idx > *size) return;

      int temp = 0;
      int before = (idx + 1) % *size;
      int after = idx - 1;
      if (after < 0) after = *size - 1;


      temp += array[idx];
      temp += array[before];
      temp += array[after];

      __syncthreads(); // Barrera...
      array[idx] = temp;
}

extern "C" void call_device_sum(int *h_a, int size)
{
      int *dev_a = NULL;
      int *dev_size = NULL;

      hipMalloc(&dev_size, sizeof(int));
      hipMemset(dev_size, 0, sizeof(int));
      hipMemcpy(dev_size, &size, sizeof(int), hipMemcpyHostToDevice);

      hipMalloc(&dev_a, (size + 2) * sizeof(int));
      hipMemset(dev_a, 0, (size + 2) * sizeof(int));
      hipMemcpy(dev_a, h_a, (size + 2) * sizeof(int), hipMemcpyHostToDevice);

      __add__ <<<BLOCKS, THREADS>>>(dev_a, dev_size);

      // se transfieren los datos del dispositivo a memoria.
      hipMemcpy(h_a, dev_a, (size + 2) * sizeof(int), hipMemcpyDeviceToHost);
      hipFree(dev_a);
      hipFree(dev_size);
}
